#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <ATen/ATen.h>
#include <ATen/native/cuda/MemoryAccess.cuh>
#include <ATen/cuda/HIPContext.h>

using namespace at::native::memory;
__managed__ double4 buffer1[1024];
__managed__ double4 buffer2[1024];

void reset_buffers() {
  for (int i = 0; i < 1024; i++) {
    buffer1[i].x = i;
    buffer1[i].y = i + 0.1;
    buffer1[i].z = i + 0.2;
    buffer1[i].w = i + 0.3;

    buffer2[2].x = -i;
    buffer2[2].y = -(i + 0.1);
    buffer2[2].z = -(i + 0.2);
    buffer2[2].w = -(i + 0.3);
  }
}

TEST(TestVectorizedMemoryAccess, CanVectorizeUpTo) {
  char *ptr = reinterpret_cast<char *>(buffer1);

  ASSERT_EQ(can_vectorize_up_to<bool>(ptr), 4);
  ASSERT_EQ(can_vectorize_up_to<int8_t>(ptr), 4);
  ASSERT_EQ(can_vectorize_up_to<int16_t>(ptr), 4);
  ASSERT_EQ(can_vectorize_up_to<int>(ptr), 4);
  ASSERT_EQ(can_vectorize_up_to<int64_t>(ptr), 4);

  ASSERT_EQ(can_vectorize_up_to<bool>(ptr + 1), 1);
  ASSERT_EQ(can_vectorize_up_to<int8_t>(ptr + 1), 1);

  ASSERT_EQ(can_vectorize_up_to<bool>(ptr + 2), 2);
  ASSERT_EQ(can_vectorize_up_to<int8_t>(ptr + 2), 2);
  ASSERT_EQ(can_vectorize_up_to<int16_t>(ptr + 2), 1);

  ASSERT_EQ(can_vectorize_up_to<bool>(ptr + 4), 4);
  ASSERT_EQ(can_vectorize_up_to<int8_t>(ptr + 4), 4);
  ASSERT_EQ(can_vectorize_up_to<int16_t>(ptr + 4), 2);
  ASSERT_EQ(can_vectorize_up_to<int>(ptr + 4), 1);

  ASSERT_EQ(can_vectorize_up_to<bool>(ptr + 8), 4);
  ASSERT_EQ(can_vectorize_up_to<int8_t>(ptr + 8), 4);
  ASSERT_EQ(can_vectorize_up_to<int16_t>(ptr + 8), 4);
  ASSERT_EQ(can_vectorize_up_to<int>(ptr + 8), 2);
  ASSERT_EQ(can_vectorize_up_to<int64_t>(ptr + 8), 1);
}

// The following kernel copy values by using vectorized policies
// defined in `ATen/native/cuda/MemoryAccess.cuh`
template <typename scalar_t, int vec_size>
__global__ void vectorized_copy(scalar_t *dst, scalar_t *src) {
  using vectorized = policies<64, 4>::vectorized<vec_size>;
  auto policy = vectorized();
  scalar_t buf[vectorized::thread_work_size];
  auto accessor = [&](int index) -> scalar_t & { return buf[index]; };
  policy.load(accessor, src + 256 * blockIdx.x);
  policy.store(accessor, dst + 256 * blockIdx.x);
}

TEST(TestVectorizedMemoryAccess, CopyKernel) {
  if (!at::cuda::is_available()) {
    return;
  }

  double *b1 = reinterpret_cast<double *>(buffer1);
  double *b2 = reinterpret_cast<double *>(buffer2);

  // vec4 copy
  reset_buffers();
  hipDeviceSynchronize();
  vectorized_copy<double, 4><<<16, 64>>>(b2, b1);
  hipDeviceSynchronize();
  ASSERT_EQ(hipGetLastError(), hipSuccess);
  for (int i = 0; i < 1024; i++) {
    ASSERT_EQ(buffer1[i].x, buffer2[i].x);
    ASSERT_EQ(buffer1[i].y, buffer2[i].y);
    ASSERT_EQ(buffer1[i].z, buffer2[i].z);
    ASSERT_EQ(buffer1[i].w, buffer2[i].w);
  }

  // vec2 copy
  reset_buffers();
  hipDeviceSynchronize();
  vectorized_copy<double, 2><<<16, 64>>>(b2, b1);
  hipDeviceSynchronize();
  ASSERT_EQ(hipGetLastError(), hipSuccess);
  for (int i = 0; i < 1024; i++) {
    ASSERT_EQ(buffer1[i].x, buffer2[i].x);
    ASSERT_EQ(buffer1[i].y, buffer2[i].y);
    ASSERT_EQ(buffer1[i].z, buffer2[i].z);
    ASSERT_EQ(buffer1[i].w, buffer2[i].w);
  }

  // vec1 copy
  reset_buffers();
  hipDeviceSynchronize();
  vectorized_copy<double, 1><<<16, 64>>>(b2, b1);
  hipDeviceSynchronize();
  ASSERT_EQ(hipGetLastError(), hipSuccess);
  for (int i = 0; i < 1024; i++) {
    ASSERT_EQ(buffer1[i].x, buffer2[i].x);
    ASSERT_EQ(buffer1[i].y, buffer2[i].y);
    ASSERT_EQ(buffer1[i].z, buffer2[i].z);
    ASSERT_EQ(buffer1[i].w, buffer2[i].w);
  }

  // unaligned
  for (int i = 0; i < 16; i++) {
    for (int j = 0; j < 16; j++) {
      b1 = reinterpret_cast<double *>(reinterpret_cast<char *>(buffer1) + i);
      b2 = reinterpret_cast<double *>(reinterpret_cast<char *>(buffer2) + j);
      hipGetLastError();
      hipDeviceSynchronize();
      vectorized_copy<double, 4><<<1, 64>>>(b2, b1);
      hipDeviceSynchronize();
      auto err = hipGetLastError();
      if (i % 16 == 0 && j % 16 == 0) {
        ASSERT_EQ(err, hipSuccess);
      } else {
        ASSERT_EQ(err, cudaErrorMisalignedAddress);
      }
    }
  }
}
